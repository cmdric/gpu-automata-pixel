
#include "kernelInvoker.cuh"
#include "kernel.cuh"


extern int* h_no_sensors;
extern int* h_no_hits;
extern int* h_sensor_Zs;
extern int* h_sensor_hitStarts;
extern int* h_sensor_hitNums;
extern int* h_hit_IDs;
extern float* h_hit_Xs;
extern float* h_hit_Ys;
extern int* h_hit_Zs;

#define cudaCheck(stmt) do {										\
        hipError_t err = stmt;										\
        if (err != hipSuccess) {									\
            std::cerr << "Failed to run " << #stmt << std::endl;    \
            return err;										        \
        }															\
    } while(0)

// Helper function for using CUDA to add vectors in parallel.
hipError_t invokeParallelSearch(dim3 numBlocks, dim3 numThreads,
	char* input, int size, Track*& tracks, int*& num_tracks, int*& h_track_indexes){
    
	// int* h_prevs, *h_nexts;
	// Histo histo;

	char *dev_input = 0;
	int* dev_num_tracks = 0;
	int* dev_track_indexes = 0;
	Track *dev_tracks = 0;
	bool* dev_track_holders = 0;
	int* dev_prevs = 0;
	int* dev_nexts = 0;
	int* dev_tracks_to_process = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaCheck(hipSetDevice(0));
    
	// Allocate memory
	// Allocate CPU buffers
	tracks = (Track*) malloc(MAX_TRACKS * sizeof(Track));
	num_tracks = (int*) malloc(sizeof(int));

	int* h_prevs = (int*) malloc(h_no_hits[0] * sizeof(int));
	int* h_nexts = (int*) malloc(h_no_hits[0] * sizeof(int));
	bool* h_track_holders = (bool*) malloc(MAX_TRACKS * sizeof(bool));
	h_track_indexes = (int*) malloc(MAX_TRACKS * sizeof(int));

    // Allocate GPU buffers
    cudaCheck(hipMalloc((void**)&dev_tracks, MAX_TRACKS * sizeof(Track)));
	cudaCheck(hipMalloc((void**)&dev_track_holders, MAX_TRACKS * sizeof(bool)));
	cudaCheck(hipMalloc((void**)&dev_track_indexes, MAX_TRACKS * sizeof(int)));
	cudaCheck(hipMalloc((void**)&dev_tracks_to_process, MAX_TRACKS * sizeof(int)));

	cudaCheck(hipMalloc((void**)&dev_prevs, h_no_hits[0] * sizeof(int)));
	cudaCheck(hipMalloc((void**)&dev_nexts, h_no_hits[0] * sizeof(int)));
    
    // Copy input file from host memory to GPU buffers
    cudaCheck(hipMalloc((void**)&dev_input, size));
    cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));
    
	// memcpys
    cudaCheck(hipMemcpy(dev_input, input, size, hipMemcpyHostToDevice));

	// Launch a kernel on the GPU with one thread for each element.
	prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts, dev_track_holders);

	// gpuKalman
	gpuKalman<<<46, 32>>>(dev_tracks, dev_track_holders);
	
	cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
	for(int i=0; i<h_no_hits[0]; ++i){
		if(h_track_holders[i]){
			printTrack(tracks, i);
		}
	}
	// histo.plotChi2("after-kalman.root", h_track_holders, tracks, h_no_hits[0]);

	postProcess<<<1, 32>>>(dev_tracks, dev_track_holders, dev_track_indexes, dev_num_tracks, dev_tracks_to_process);
	cudaCheck(hipMemcpy(h_track_indexes, dev_track_indexes, MAX_TRACKS * sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	std::cout << std::endl
			  << "Post-processed:" << std::endl;
	for(int i=0; i<num_tracks[0]; ++i){
		printTrack(tracks, h_track_indexes[i]);
	}
	std::cout << std::endl << "Generated " << num_tracks[0] << " tracks" << std::endl;

	// histo.plotChi2("after-post-processing.root", h_track_indexes, tracks, num_tracks[0]);

    // neighboursFinder<<<numBlocks, numThreads>>>();

	// Visualize results
	/*cudaCheck(hipMemcpy(h_prevs, dev_prevs, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_nexts, dev_nexts, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	printOutSensorHits(2, h_prevs, h_nexts);*/

	/*
	out = std::ofstream("prevnexts.out");
	out.write((char*) &h_prevs[0], h_no_hits[0] * sizeof(int));
	out.write((char*) &h_nexts[0], h_no_hits[0] * sizeof(int));
	out.close();
	*/

	// neighboursCleaner<<<numBlocks, numThreads>>>();
	
	// Visualize results
	/*cudaCheck(hipMemcpy(h_prevs, dev_prevs, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_nexts, dev_nexts, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	// printOutSensorHits(2, h_prevs, h_nexts);
	printOutAllSensorHits(h_prevs, h_nexts);*/
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaCheck(hipDeviceSynchronize());
    
	// cuda copy back
	cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	
    // Copy output vector from GPU buffer to host memory.
    cudaCheck(hipMemcpy(tracks, dev_tracks, num_tracks[0] * sizeof(Track), hipMemcpyDeviceToHost));
    
    return cudaStatus;
}

// #track, h0, h1, h2, h3, ..., hn, length, chi2
void printTrack(Track* tracks, int track_no){
	std::cout << track_no << ": ";

	Track t = tracks[track_no];
	for(int i=0; i<t.hitsNum; ++i){
		std::cout << h_hit_IDs[t.hits[i]] << ", ";
	}

	std::cout << "length: " << (int) t.hitsNum << std::endl;
}

/*
float f_chi2(Track& t)
{
	float ch = 0.0;
	int nDoF  = -4;
	int hitNumber;
	for (int i=0; i<t.hitsNum; ++i){
		hitNumber = t.hits[i];
		ch += f_chi2Track(t, hitNumber);
		nDoF += 2;
	}
	return ch/nDoF;
}
*/

void printOutAllSensorHits(int* prevs, int* nexts){
	std::cout << "All valid sensor hits: " << std::endl;
	for(int i=0; i<h_no_sensors[0]; ++i){
		for(int j=0; j<h_sensor_hitNums[i]; ++j){
			int hit = h_sensor_hitStarts[i] + j;
			
			if(nexts[hit] != -1){
				std::cout << hit << ", " << nexts[hit] << std::endl;
			}
		}
	}
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts){
	for(int i=0; i<h_sensor_hitNums[sensorNumber]; ++i){
		int hstart = h_sensor_hitStarts[sensorNumber];

		std::cout << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
	}
}

void getMaxNumberOfHits(char*& input, int& maxHits){
	int* l_no_sensors = (int*) &input[0];
    int* l_no_hits = (int*) (l_no_sensors + 1);
    int* l_sensor_Zs = (int*) (l_no_hits + 1);
    int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
    int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

	maxHits = 0;
	for(int i=0; i<l_no_sensors[0]; ++i){
		if(l_sensor_hitNums[i] > maxHits)
			maxHits = l_sensor_hitNums[i];
	}
}
