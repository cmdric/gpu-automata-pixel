#include "hip/hip_runtime.h"
﻿
#include "kernel.cuh"

__global__ void prepareData(char* input, int* _prevs, int* _nexts, bool* track_holders){
	no_sensors = (int*) &input[0];
    no_hits = (int*) (no_sensors + 1);
    sensor_Zs = (int*) (no_hits + 1);
    sensor_hitStarts = (int*) (sensor_Zs + no_sensors[0]);
    sensor_hitNums = (int*) (sensor_hitStarts + no_sensors[0]);
    hit_IDs = (int*) (sensor_hitNums + no_sensors[0]);
    hit_Xs = (float*) (hit_IDs + no_hits[0]);
	hit_Ys = (float*) (hit_Xs + no_hits[0]);
	hit_Zs = (int*) (hit_Ys + no_hits[0]);

	prevs = _prevs;
	nexts = _nexts;

	for(int i=0; i<MAX_TRACKS; ++i){
		track_holders[i] = false;
	}
}

__global__ void neighboursFinder()
{
	__shared__ Hit prev_hits[HITS_SHARED];
	__shared__ Hit next_hits[HITS_SHARED];
	__shared__ Sensor s[3];

	/*
	gridDim.{x,y,z}
	blockIdx.{x,y,z}

	blockDim.{x,y,z}
	threadIdx.{x,y,z}
	*/

	int current_sensor, prev_sensor, next_sensor, address;

	current_sensor = blockIdx.x;
	prev_sensor = current_sensor - 2;
	next_sensor = current_sensor + 2;

	// Prepare input
	if (threadIdx.x == 0 || threadIdx.x == 1 || threadIdx.x == 2){
		
		// trick to execute things in the same warp
		bool condition = (prev_sensor >= 0 && threadIdx.x == 0) || (next_sensor <= 48 && threadIdx.x == 2) || threadIdx.x == 1;
		if (condition){
			address = prev_sensor * (threadIdx.x==0) +
				current_sensor * (threadIdx.x==1) + next_sensor * (threadIdx.x==2);
			s[threadIdx.x].z = sensor_Zs[address];
			s[threadIdx.x].hitStart = sensor_hitStarts[address];
			s[threadIdx.x].hitNums = sensor_hitNums[address];
		}
	}

	__syncthreads();

	Hit current_hit;
	float best_fit;
	int best_prev;
	int best_next;
	int current_element;
	int next_num_hits_to_load;
	int prev_num_hits_to_load;
	int current_num_hits_to_load;
	int prev_element;
	int next_element;
	float fit, t, x, y, d1, d2;
	bool fit_is_better;

	// TODO: Account for special cases (2 first sensors, and 2 last sensors)
	// if (prev_sensor < 0 || next_sensor > NUM_SENSORS){
	if(false){
		if (next_sensor > NUM_SENSORS){
			current_sensor -= 2;
			next_sensor -= 2;
		}

		current_num_hits_to_load = int(ceilf(s[1].hitNums / blockDim.x));
		next_num_hits_to_load = int(ceilf(s[2].hitNums / blockDim.x));
		
		best_fit = MAX_FLOAT;
		best_prev = -1;
		best_next = -1;
		
		// Load elements into
		// - current_hit: The element we are treating
		// - prev_hits:   Previous hits (HITS_SHARED === blockDim.x)
		// - next_hits:   Next hits (HITS_SHARED === blockDim.x)

		for (int i=0; i<current_num_hits_to_load; ++i){
			current_element = i * blockDim.x + threadIdx.x;

			if (current_element < s[1].hitNums){
				current_hit.x = hit_Xs[s[1].hitStart + current_element];
				current_hit.y = hit_Ys[s[1].hitStart + current_element];
			}

			for (int k=0; k<next_num_hits_to_load; ++k){
				next_element = k * blockDim.x + threadIdx.x;

				if (next_element < s[2].hitNums){
					next_hits[threadIdx.x].x = hit_Xs[s[2].hitStart + next_element];
					next_hits[threadIdx.x].y = hit_Ys[s[2].hitStart + next_element];
				}

				// Start comparison, minimize the best_fit for each current_element
				if(current_element < s[1].hitNums){
					// Minimize best fit
					for (int m=0; m<HITS_SHARED; ++m){
						// float fit;
						// fit = prev_hits[m]

						/* Special cases calculation
						d is h0-h1 distance to <0,0,0> on plane s0.
						*/

						t = - s[1].z / s[2].z - s[1].z;
						x = current_hit.x + t * (next_hits[m].x - current_hit.x);
						y = current_hit.y + t * (next_hits[m].y - current_hit.y);
						fit = powf( (float) (x), 2.0) + 
							  powf( (float) (y), 2.0);
								
						fit_is_better = fit < best_fit;
						best_fit = fit_is_better * fit + !fit_is_better * best_fit;
						best_next = fit_is_better * (s[2].hitStart + k * blockDim.x + m) + 
									!fit_is_better * best_next;
					}
				}
			}
		}
		
		// Store best fit into solution array.
		if(prev_sensor < 0){
			nexts[s[1].hitStart + current_element] = best_next;
		}
		else {
			prevs[s[1].hitStart + current_element] = best_next;
		}
	}

	if (prev_sensor >= 0 && next_sensor <= NUM_SENSORS){

		prev_num_hits_to_load = int(ceilf(s[0].hitNums / blockDim.x));
		current_num_hits_to_load = int(ceilf(s[1].hitNums / blockDim.x));
		next_num_hits_to_load = int(ceilf(s[2].hitNums / blockDim.x));
		
		best_fit = MAX_FLOAT;
		best_prev = -1;
		best_next = -1;

		// Load elements into
		// - current_hit: The element we are treating
		// - prev_hits:   Previous hits (HITS_SHARED === blockDim.x)
		// - next_hits:   Next hits (HITS_SHARED === blockDim.x)

		for (int i=0; i<current_num_hits_to_load; ++i){
			current_element = i * blockDim.x + threadIdx.x;

			if (current_element < s[1].hitNums){
				current_hit.x = hit_Xs[s[1].hitStart + current_element];
				current_hit.y = hit_Ys[s[1].hitStart + current_element];
			}

			for (int j=0; j<prev_num_hits_to_load; ++j){
				prev_element = j * blockDim.x + threadIdx.x;

				if (prev_element < s[0].hitNums){
					prev_hits[threadIdx.x].x = hit_Xs[s[0].hitStart + prev_element];
					prev_hits[threadIdx.x].y = hit_Ys[s[0].hitStart + prev_element];
				}

				for (int k=0; k<next_num_hits_to_load; ++k){
					next_element = k * blockDim.x + threadIdx.x;

					if (next_element < s[2].hitNums){
						next_hits[threadIdx.x].x = hit_Xs[s[2].hitStart + next_element];
						next_hits[threadIdx.x].y = hit_Ys[s[2].hitStart + next_element];
					}

					// Start comparison, minimize the best_fit for each current_element
					if(current_element < s[1].hitNums){
						// Minimize best fit
						for (int m=0; m<HITS_SHARED; ++m){
							for (int n=0; n<HITS_SHARED; ++n){
								// float fit;
								// fit = prev_hits[m]

								/* Calculation of the best fit:
								hits on sensors 0, 1 and 2 are h0, h1 and h2. We are calculating
								the best h0 and h2 for h1.

								d1 is the distance from the line h0-h2 to h1 in plane sensor s1.
								d2 is the distance from the line h0-h2 to <0,0,0> in plane sensor s0.
								*/

								t = s[1].z - s[0].z / s[2].z - s[0].z;
								x = prev_hits[m].x + t * (next_hits[n].x - prev_hits[m].x);
								y = prev_hits[m].y + t * (next_hits[n].y - prev_hits[m].y);
								d1 = sqrtf( powf( (float) (current_hit.x - x), 2.0) + 
											powf( (float) (current_hit.y - y), 2.0));

								t = - s[0].z / s[2].z - s[0].z;
								x = prev_hits[m].x + t * (next_hits[n].x - prev_hits[m].x);
								y = prev_hits[m].y + t * (next_hits[n].y - prev_hits[m].y);
								d2 = sqrtf( powf( (float) (x), 2.0) + 
											powf( (float) (y), 2.0));

								// fit = powf(d1, 2.0) + d2;
								fit = d1;
								
								fit_is_better = fit < best_fit;
								best_fit = fit_is_better * fit + !fit_is_better * best_fit;
								best_prev = fit_is_better * (s[0].hitStart + j * blockDim.x + m) +
											!fit_is_better * best_prev;
								best_next = fit_is_better * (s[2].hitStart + k * blockDim.x + n) + 
											!fit_is_better * best_next;
							}
						}
					}
				}
			}

			// Store best fit into solution array.
			if(current_element < s[1].hitNums){
				prevs[s[1].hitStart + current_element] = best_prev;
				nexts[s[1].hitStart + current_element] = best_next;
			}
		}
	}
}

__global__ void neighboursCleaner()
{
	int block_size = int(ceilf( ((float)no_hits[0]) / gridDim.x));
	int thread_size = int(ceilf( ((float)block_size) / blockDim.x));

	for(int j=0; j<thread_size; ++j){
		int current_hit = blockIdx.x * block_size + blockDim.x * j + threadIdx.x;
		if(current_hit < no_hits[0]){
			int next_hit = nexts[current_hit];
			if(next_hit < 0 || prevs[next_hit] != current_hit){
				prevs[next_hit] = -1;
				nexts[current_hit] = -1;
			}
		}
	}
}

/** fitHits, gives the fit between h0 and h1.

The accept condition requires dxmax and dymax to be in a range.

The fit (d1) depends on the distance of the tracklet to <0,0,0>.
*/
__device__ float fitHits(Hit& h0, Hit& h1, Sensor& s0, Sensor& s1){
	// Max dx, dy permissible over next hit

	// TODO: This can go outside this function (only calc once per pair
	// of sensors). Also, it could only be calculated on best fitting distance d1.
	float s_dist = fabs((float)( s1.z - s0.z ));
	float dxmax = PARAM_MAXXSLOPE * s_dist;
	float dymax = PARAM_MAXYSLOPE * s_dist;
	
	bool accept_condition = fabs(h1.x - h0.x) < dxmax &&
							fabs(h1.y - h0.y) < dymax;

	/*float dxmax = PARAM_MAXXSLOPE * fabs((float)( s1.z - s0.z ));
	float dymax = PARAM_MAXYSLOPE * fabs((float)( s1.z - s0.z ));*/
	
	// Distance to <0,0,0> in its XY plane.
	/* float t = - ((float) s0.z) / ((float) (s1.z - s0.z));
	float x = h0.x + t * (h1.x - h0.x);
	float y = h0.y + t * (h1.y - h0.y);
	float d1 = powf( (float) (x), 2.0) + 
			   powf( (float) (y), 2.0); */

	// Distance between the hits on an XY projection
	float d1 = powf( (float) (h1.x - h0.x), 2.0f) +
               powf( (float) (h1.y - h0.y), 2.0f);

	return accept_condition * d1 + !accept_condition * MAX_FLOAT;
}

// TODO: Optimize with Olivier's
__device__ float fitHitToTrack(Track& t, Hit& h1, Sensor& s1){
	// tolerance
	// TODO: To improve efficiency, try with PARAM_TOLERANCE_EXTENDED
	float x_prediction = t.x0 + t.tx * s1.z;
	bool tol_condition = fabs(x_prediction - h1.x) < PARAM_TOLERANCE;

	// chi2 of hit (taken out from function for efficiency)
	float dx = x_prediction - h1.x;
	float dy = (t.y0 + t.ty * s1.z) - h1.y;
	float chi2 = dx * dx * PARAM_W + dy * dy * PARAM_W;

	// TODO: The check for chi2_condition can totally be done after this call
	bool chi2_condition = chi2 < PARAM_MAXCHI2;
	
	return tol_condition * chi2_condition * chi2 + (!tol_condition || !chi2_condition) * MAX_FLOAT;
}

// Create track
__device__ void acceptTrack(Track& t, Hit& h0, Hit& h1, Sensor& s0, Sensor& s1, int h0_num, int h1_num){
	float wz = PARAM_W * s0.z;

	t.s0 = PARAM_W;
	t.sx = PARAM_W * h0.x;
	t.sz = wz;
	t.sxz = wz * h0.x;
	t.sz2 = wz * s0.z;

	t.u0 = PARAM_W;
	t.uy = PARAM_W * h0.y;
	t.uz = wz;
	t.uyz = wz * h0.y;
	t.uz2 = wz * s0.z;

	t.hitsNum = 1;
	t.hits[0] = h0_num;

	// note: This could be done here (inlined)
	updateTrack(t, h1, s1, h1_num);
}

// Update track
__device__ void updateTrack(Track& t, Hit& h1, Sensor& s1, int h1_num){
	float wz = PARAM_W * s1.z;

	t.s0 += PARAM_W;
	t.sx += PARAM_W * h1.x;
	t.sz += wz;
	t.sxz += wz * h1.x;
	t.sz2 += wz * s1.z;

	t.u0 += PARAM_W;
	t.uy += PARAM_W * h1.y;
	t.uz += wz;
	t.uyz += wz * h1.y;
	t.uz2 += wz * s1.z;

	t.hits[t.hitsNum] = h1_num;
	t.hitsNum++;

	updateTrackCoords(t);
}

// TODO: Check this function
__device__ void updateTrackCoords (Track& t){
	float den = ( t.sz2 * t.s0 - t.sz * t.sz );
	if ( fabs(den) < 10e-10 ) den = 1.f;
	t.tx     = ( t.sxz * t.s0  - t.sx  * t.sz ) / den;
	t.x0     = ( t.sx  * t.sz2 - t.sxz * t.sz ) / den;

	den = ( t.uz2 * t.u0 - t.uz * t.uz );
	if ( fabs(den) < 10e-10 ) den = 1.f;
	t.ty     = ( t.uyz * t.u0  - t.uy  * t.uz ) / den;
	t.y0     = ( t.uy  * t.uz2 - t.uyz * t.uz ) / den;
}

/** Simple implementation of the Kalman Filter selection on the GPU (step 4).

Will rely on pre-processing for selecting next-hits for each hit.

Implementation,
- Perform implementation searching on all hits for each sensor

The algorithm has two parts:
- Tracklet creation (two hits)
- Track following (consecutive sensors)


Optimizations,
- Optimize with shared memory
- Optimize further with pre-processing

Then there must be a post-processing, which selects the
best tracks based on (as per the conversation with David):
- length
- chi2

For this, simply use the table with all created tracks (postProcess):

#track, h0, h1, h2, h3, ..., hn, length, chi2

*/

__global__ void gpuKalman(Track* tracks, bool* track_holders){
	Track t;
	Sensor s0, s1;
	Hit h0, h1;

	float fit, best_fit;
	bool fit_is_better, accept_track;
	int best_hit, current_hit;

	int current_sensor = (47 - blockIdx.x);

	s0.hitStart = sensor_hitStarts[current_sensor];
	s0.hitNums = sensor_hitNums[current_sensor];
	s0.z = sensor_Zs[current_sensor];

	// Analyze the best hit for next sensor
	int next_sensor = current_sensor - 2;

	if(next_sensor >= 0){
		// Iterate in all hits for current sensor
		for(int i=0; i<int(ceilf( ((float) s0.hitNums) / blockDim.x)); ++i){
			next_sensor = current_sensor - 2;
			current_hit = blockIdx.x * i + threadIdx.x;
			if(current_hit < s0.hitNums){

				h0.x = hit_Xs[ s0.hitStart + current_hit ];
				h0.y = hit_Ys[ s0.hitStart + current_hit ];

				// Initialize track
				for(int j=0; j<TRACK_SIZE; ++j){
					t.hits[j] = -1;
				}

				// TODO: shared memory.
				s1.hitStart = sensor_hitStarts[next_sensor];
				s1.hitNums = sensor_hitNums[next_sensor];
				s1.z = sensor_Zs[next_sensor];
		
				// TRACK CREATION
				// TODO: Modify with preprocessed list of hits.
				best_fit = MAX_FLOAT;
				best_hit = -1;
				for(int j=0; j<sensor_hitNums[next_sensor]; ++j){
					// TODO: Load in chunks of SHARED_MEMORY and take
					// them from shared memory.
					h1.x = hit_Xs[s1.hitStart + j];
					h1.y = hit_Ys[s1.hitStart + j];

					fit = fitHits(h0, h1, s0, s1);
					fit_is_better = fit < best_fit;

					best_fit = fit_is_better * fit + !fit_is_better * best_fit;
					best_hit = fit_is_better * j + !fit_is_better * best_hit;
				}

				accept_track = best_fit != MAX_FLOAT;

				// We have a best fit!

				// For those who have tracks, we go on
				if(accept_track){
					// Fill in t (ONLY in case the best fit is acceptable)
					acceptTrack(t, h0, h1, s0, s1, s0.hitStart + current_hit, s1.hitStart + best_hit);

					// TRACK FOLLOWING
					next_sensor -= 2;
					while(next_sensor >= 0){
						// Go to following sensor
						/*s0.hitNums = s1.hitNums;
						s0.hitStart = s1.hitStart;
						s0.z = s1.z;*/
						
						s1.hitStart = sensor_hitStarts[next_sensor];
						s1.hitNums = sensor_hitNums[next_sensor];
						s1.z = sensor_Zs[next_sensor];

						best_fit = MAX_FLOAT;
						for(int k=0; k<sensor_hitNums[next_sensor]; ++k){
							// TODO: Load in chunks of SHARED_MEMORY and take
							// them from shared memory.
							h1.x = hit_Xs[s1.hitStart + k];
							h1.y = hit_Ys[s1.hitStart + k];

							fit = fitHitToTrack(t, h1, s1);
							fit_is_better = fit < best_fit;

							best_fit = fit_is_better * fit + !fit_is_better * best_fit;
							best_hit = fit_is_better * k + !fit_is_better * best_hit;
						}

						// We have a best fit!
						// Fill in t, ONLY in case the best fit is acceptable

						// TODO: Maybe try to do this more "parallel"
						if(best_fit != MAX_FLOAT){
							updateTrack(t, h1, s1, s1.hitStart + best_hit);
						}

						next_sensor -= 2;
					}
				}

				// If it's a track, write it to memory, as long as 
				// it fulfills a minimum length.
				track_holders[s0.hitStart + current_hit] = accept_track && (t.hitsNum >= MIN_HITS_TRACK);
				if(accept_track && (t.hitsNum >= MIN_HITS_TRACK)){
					tracks[s0.hitStart + current_hit] = t;
				}
			}
		}
	}
}


/* Calculating the chi2 of a track is quite cumbersome.
It implies loading hit_Xs, hit_Ys, and sensor_Zs elements for each
hit of the track. This introduces branching, and is slow.

However, the track chi2 has to be calculated only when the
track has been created (the tx, ty values change).
*/

__device__ float trackChi2(Track& t){
	float ch = 0.0;
	int nDoF  = -4 + 2 * t.hitsNum;
	Hit h;
	for (int i=0; i<TRACK_SIZE; i++){
		// TODO: Maybe there's a better way to do this
		if(t.hits[i] != -1){
			h.x = hit_Xs[ t.hits[i] ];
			h.y = hit_Ys[ t.hits[i] ];

			ch += hitChi2(t, h, hit_Zs[ t.hits[i] ]);
		}
	}
	return ch/nDoF;
}

__device__ float hitChi2(Track& t, Hit& h, int hit_z){
	// chi2 of a hit
	float dx = (t.x0 + t.tx * hit_z) - h.x;
	float dy = (t.y0 + t.ty * hit_z) - h.y;
	return dx * dx * PARAM_W + dy * dy * PARAM_W;
}


/** The postProcess method takes care of discarding tracks
which are redundant. In other words, it will (hopefully) increase
the purity of our tracks.

- Inspect track_holders and generate track_indexes and num_tracks

The main idea is to accept tracks which have unique (> REQUIRED_UNIQUES) hits.
For this, each track is checked against all other more preferent tracks, and
non common hits are kept.

TODO: Change preference system by something more civilized.
A track t0 has preference over another t1 one if:
t0.hitsNum > t1.hitsNum ||
(t0.hitsNum == t1.hitsNum && chi2(t0) < chi2(t1))
*/
__global__ void postProcess(Track* tracks, bool* track_holders, int* track_indexes, int* num_tracks, int* tracks_to_process){
	// tracks_to_process holds the list of tracks with track_holders[t] == true
	
	// TODO: Try with sh_tracks_to_process
	// __shared__ int sh_tracks_to_process[MAX_POST_TRACKS];

	__shared__ Track sh_tracks[BUNCH_POST_TRACKS];
	__shared__ float sh_chi2[BUNCH_POST_TRACKS];

	__shared__ Track sh_next_tracks[BUNCH_POST_TRACKS];
	__shared__ float sh_next_chi2[BUNCH_POST_TRACKS];
	
	// We will use an atomic to write on a vector concurrently on several values
	__shared__ int tracks_to_process_size;
	__shared__ int tracks_accepted_size;

	tracks_to_process_size = 0;
	tracks_accepted_size = 0;

	__syncthreads(); // for the atomics tracks_to_process_size, and tracks_processed

	int i, j, current_track, next_track;
	bool preferent;

	for(i=0; i<int(ceilf( ((float) no_hits[0]) / blockDim.x)); ++i){
		current_track = blockDim.x * i + threadIdx.x;
		if(current_track < no_hits[0]){
			// Iterate in all tracks (current_track)

			if(track_holders[current_track]){
				// Atomic add
				int current_atomic = atomicAdd(&tracks_to_process_size, 1);

				// TODO: This condition shouldn't exist,
				// redo using method to process in batches if necessary
				// if(current_atomic < MAX_POST_TRACKS)
				tracks_to_process[current_atomic] = current_track;
			}
		}
	}

	__syncthreads();

	// Iterate in all current_tracks against all next_tracks.
	// Do this processing on batches of blockDim.x size
	for(i=0; i<int(ceilf( ((float) tracks_to_process_size) / blockDim.x)); ++i){
		current_track = blockDim.x * i + threadIdx.x;
		if(current_track < tracks_to_process_size){
			// Store all tracks in sh_tracks
			sh_tracks[threadIdx.x] = tracks[tracks_to_process[current_track]];

			// Calculate chi2
			sh_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
		}

		__syncthreads();

		// if(sh_tracks[threadIdx.x].hits[0] == 987)
		// 	i = 20;

		// Iterate in all next_tracks
		for(j=0; j<int(ceilf( ((float) tracks_to_process_size) / blockDim.x)); ++j){
			next_track = blockDim.x * j + threadIdx.x;

			if(next_track < tracks_to_process_size){
				// Store all tracks in sh_tracks
				sh_next_tracks[threadIdx.x] = tracks[tracks_to_process[next_track]];

				// Calculate chi2
				sh_next_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
			}

			__syncthreads();

			// All is loaded, commencing assault!
			for(int k=0; k<BUNCH_POST_TRACKS; ++k){
				next_track = blockDim.x * j + k;

				if(current_track < tracks_to_process_size && next_track < tracks_to_process_size){
					/* Compare all tracks to check uniqueness, based on
					- length
					- chi2

					preferent is a boolean storing this logic. It reads,
				
					TODO: Change preference system by something more civilized
					next_track is preferent if
						it's not current_track,
						its length > current_track . length OR
						(its length == current_track . length AND
						chi2 < current_track . chi2)
					*/
					preferent = current_track!=next_track &&
										(sh_next_tracks[k].hitsNum > sh_tracks[threadIdx.x].hitsNum ||
										(sh_next_tracks[k].hitsNum == sh_tracks[threadIdx.x].hitsNum &&
										sh_next_chi2[k] < sh_chi2[threadIdx.x]));

					// TODO: Maybe there's a better way...
					if(preferent){
						// Eliminate hits from current_track, based on next_track's
						for(int current_hit=0; current_hit<TRACK_SIZE; ++current_hit){
							for(int next_hit=0; next_hit<TRACK_SIZE; ++next_hit){
								/* apply mask:
								a[i] = 
									(a[i] == b[j]) * -1 +
									(a[i] != b[j]) * a[i]
								*/
								sh_tracks[threadIdx.x].hits[current_hit] =
									(sh_tracks[threadIdx.x].hits[current_hit] == sh_next_tracks[k].hits[next_hit]) * -1 + 
									(sh_tracks[threadIdx.x].hits[current_hit] != sh_next_tracks[k].hits[next_hit]) *
										sh_tracks[threadIdx.x].hits[current_hit];
							}
						}
					}
				}
			}
		}

		if(current_track < tracks_to_process_size){
			// Check how many uniques do we have
			int unique = 0;
			for(int hit=0; hit<TRACK_SIZE; ++hit)
				unique += (sh_tracks[threadIdx.x].hits[hit]!=-1);

			if(!ALLOW_POSTPROCESSING || ((float) unique) / sh_tracks[threadIdx.x].hitsNum > REQUIRED_UNIQUES){
				int current_track_accepted = atomicAdd(&tracks_accepted_size, 1);

				track_indexes[current_track_accepted] = tracks_to_process[current_track];
			}
		}
	}

	__syncthreads();

	if(threadIdx.x==0)
		num_tracks[0] = tracks_accepted_size;
}
